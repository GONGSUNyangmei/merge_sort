
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>
#include <unistd.h>
#include <string>
#include <fcntl.h>
#include <random>

using namespace std;

int write_disk(int in, void* tmp_buffer , unsigned long long size , unsigned long long file_offset)
{
    unsigned long long left_size = size;
    while(left_size > 0) {
        ssize_t return_value;
        return_value = pwrite(in, tmp_buffer, size, file_offset);
        if(return_value < 0)
            return return_value;
        left_size -= return_value;
        file_offset += return_value;
    }
    return size;
}

int check_result(int fd,unsigned long long entry_num )
{
    std::cout << "check result" << std::endl;
    
    // for(int i = 0; i < 10; i++) {
    //     pread(fd, &value1, sizeof(int), i * sizeof(int));
    //     std::cout << "fd [ "<< i << "] = "<<value1 <<std::endl;
    // }
    unsigned long long tmp;
    pread(fd, &tmp, sizeof(int), (1742863087UL)*sizeof(int));
    cout<< "value [ 1742863087]  "<<tmp<<endl;
    std::mt19937 generator;
    for(unsigned long long i = 1; i < 1000; i++) {
        unsigned long long value1;
        unsigned long long value2;
        unsigned long long index1 = generator()%(entry_num -1 ); 
        unsigned long long index2 = generator()%(index1 ); 
        pread(fd, &value1, sizeof(int), (index1)*sizeof(int));
        pread(fd, &value2, sizeof(int), (index2)*sizeof(int));
        if(value2 > value1) {
            cout << "error" << endl;
            std::cout << "value [ "<<index1 <<"] = " << value1 <<std::endl;
            std::cout << "value [ "<<index2   <<"] = " << value2 <<std::endl;
            return -1;
        }
        value1 = value2;
    }
    std::cout << " =========== check pass ============ "<< std::endl;
    return 0;
}




void block_sort(int in,  unsigned long long file_offset , unsigned long long block_size, int thread_num) {
    // read data from input file
    
    int* tmp_buffer = (int * )malloc(block_size * sizeof(int));
    std::cout << " block_size  :"<< block_size  <<std::endl;
    std::cout << " file_offset :"<< file_offset << std::endl;
    pread(in, tmp_buffer, block_size * sizeof(int), file_offset);
    //void *devPtr;
    //cudaMalloc(&devPtr, block_size * sizeof(int));
    //cudaMemcpy(devPtr, tmp_buffer, block_size * sizeof(int), cudaMemcpyHostToDevice);
    // sort data in memory
    ssize_t return_value;
    for(int i = 0; i < block_size; i++) {
        tmp_buffer[i] = i/1024;
    }
    return_value = write_disk(in, tmp_buffer, block_size * sizeof(int), file_offset);
    std::cout << " write size : " << block_size * sizeof(int) << endl;
    std::cout << " pwrite return value : "<< return_value<<endl;
    for(int i = 0; i < block_size; i++) {
        int  tmp;
        return_value = pread(in, &tmp, sizeof(int), i*sizeof(int));
        if(return_value != sizeof(int))
            std::cout << " pread return value : "<< return_value<<endl;
        if(tmp_buffer[i]!=tmp)
        {
            cout<< "start enequal!!! at "<<i<<endl;
            break;
        }
        
    }
    // cout<< "value [ 1742863087]  "<<tmp_buffer[1742863087]<<endl;
    // // write sorted data to output file
    // //cudaMemcpy(tmp_buffer, devPtr, block_size * sizeof(int), cudaMemcpyDeviceToHost);
    
    

    // cout<< "value [ 1742863087]  "<<tmp<<endl;
    free(tmp_buffer);
    check_result(in,2UL*1024*1024*1024);
}

//* this function use binary search to find the partition index in the second file
void binary_find_partition_index(int in, int partitial_num, int* partitial_value, int offset, int entry_num , int * partitial_index, int* partitial_num2){
    int cursor_num = partitial_num*2  ;
    int* cursor_value = (int*)malloc(cursor_num * sizeof(int));
    int* cursor_index = (int*)malloc(cursor_num * sizeof(int));
    for(int i = 0; i < cursor_num ; i++) {
        cursor_index[i] = i*entry_num/cursor_num;
        pread(in, &cursor_value[i], sizeof(int), offset + cursor_index[i] * sizeof(int));
    }
    //! need to be checked
    int left = 0;
    int right = entry_num - 1;
    for(int i = 0; i < partitial_num -1; i++) {  // (partitial_num -1 ) partition points
        for(int j = 0; j < cursor_num; j++) {
            if(partitial_value[i] <= cursor_value[j]) {
                right = cursor_index[j];
                if(i >0 && partitial_index[i-1] > left) {
                    left = partitial_index[i-1];
                }
                else {
                    left = cursor_index[j - 1];
                }
                break;
            }
        }
        while(left < right) {
            int mid = (left + right) / 2;
            int mid_value ;
            pread(in, &mid_value, sizeof(int), offset + mid * sizeof(int));
            if(partitial_value[i] <= mid_value) {
                right = mid;
            }
            else {
                left = mid + 1;
            }
        }
        partitial_index[i] =  left;
        if(i != 0) {
            partitial_num2[i-1] = partitial_index[i] - partitial_index[i-1];
        }

        if(i == partitial_num - 1) {
            partitial_num2[i] = entry_num - partitial_index[i];
        }
        
    }
}

__global__
void merge_kernel(int** a, int** b, int** c, int* left_num_1, int* left_num_2, int* dest_num)
{
    int block_id = blockIdx.x; //todo need to be checked
    //int start_offset = blockIdx.x * blockDim.x + threadIdx.x;  //todo need to be checked
    int index_1 =0;
    int index_2 =0;
    int index_3 =0;
    while(1){
        if(a[block_id][index_1] <= b[block_id][index_2]) {
            c[block_id][index_3] = a[block_id][index_1];
            index_1++;
            index_3++;
        }
        else {
            c[block_id][index_3] = b[block_id][index_2];
            index_2++;
            index_3++;
        }
        //* judge when to escape : if a or b is empty
        if(index_1 == left_num_1[block_id] || index_2 == left_num_2[block_id] ) {
            //* move the rest of the data to the front
            if(index_1 == left_num_1[block_id]) {
                for(int i = 0; i < left_num_2[block_id] - index_2; i++) {
                    b[block_id][i] = b[block_id][i+ index_2];
                }
            }
            else {
                for(int i = 0; i < left_num_1[block_id] - index_1; i++) {
                    a[block_id][i ] = a[block_id][i+ index_1];
                }
            }
            break;
        }
    }
    left_num_1[block_id] -= index_1;
    left_num_2[block_id] -= index_2;
    dest_num[block_id] = index_3;
}

void parallel_merge(int in, int  out, int fetch_num, int thread_num, int* partitial_index1,int* partitial_index2,int* partitial_num1 ,int* partitial_num2 ,int partitial_num,int offset ) {
    int ** p_tmpbuffer1 = (int**)malloc(partitial_num * sizeof(int*));
    for(int i = 0; i < partitial_num; i++) {
        p_tmpbuffer1[i] = (int*)malloc(fetch_num * sizeof(int));
    }

    int ** p_tmpbuffer2 = (int**)malloc(partitial_num * sizeof(int*));
    for(int i = 0; i < partitial_num; i++) {
        p_tmpbuffer2[i] = (int*)malloc(fetch_num * sizeof(int));
    }
    int ** p_destbuffer = (int**)malloc(  partitial_num * sizeof(int*));
    for(int i = 0; i <  partitial_num; i++) {
        p_destbuffer[i] = (int*)malloc(2*fetch_num * sizeof(int));
    }
    int * left_num_1 = (int*)malloc(partitial_num * sizeof(int));
    for(int i = 0; i < partitial_num; i++) {
        left_num_1[i] = partitial_num1[i];
    }
    int * left_num_2 = (int*)malloc(partitial_num * sizeof(int));
    for(int i = 0; i < partitial_num; i++) {
        left_num_2[i] = partitial_num2[i];
    }
    int * tmpbuffer1_left = (int*)malloc(partitial_num * sizeof(int));
    for(int i = 0; i < partitial_num; i++) {
        tmpbuffer1_left[i] = 0;
    }
    int * tmpbuffer2_left = (int*)malloc(partitial_num * sizeof(int));
    for(int i = 0; i < partitial_num; i++) {
        tmpbuffer2_left[i] = 0;
    }
    int * destbuffer_num = (int*)malloc(partitial_num * sizeof(int));
    for(int i = 0; ; i++) { // merge two block
        bool flag = false;
        for(int j = 0; j < partitial_num; j++) {
            if(left_num_1[j] > 0 || left_num_2[j] > 0 || tmpbuffer1_left[j] > 0 || tmpbuffer2_left[j] > 0) {
                flag = true;
            }
        }
        if(!flag) {
            break;
        }
        //* phase 1: read data from input file to each tmp buffer
        for(int j = 0; j < partitial_num; j++) {
            if(left_num_1[j] > 0 && tmpbuffer1_left[j] == 0) {
                int read_num = (left_num_1[j] > fetch_num) ? fetch_num : left_num_1[j];
                pread(in, p_tmpbuffer1[j], read_num * sizeof(int), offset + partitial_index1[j] * sizeof(int));
                left_num_1[j] -= read_num;
                tmpbuffer1_left[j] = read_num;
            }
            if(left_num_2[j] > 0 && tmpbuffer2_left[j] == 0) {
                int read_num = (left_num_2[j] > fetch_num) ? fetch_num : left_num_2[j];
                pread(in, p_tmpbuffer2[j], read_num * sizeof(int), offset + partitial_index2[j] * sizeof(int));
                left_num_2[j] -= read_num;
                tmpbuffer2_left[j] = read_num;
            }
        }
        //* phase 2: use gpu kernels parallel merge data in memory and move the unsorted data from back to front
        
        int** devPtr1;
        int** devPtr2;
        int** devPtr3;
        for(int j = 0; j < partitial_num; j++) {
            hipMalloc(&devPtr1[j], fetch_num * sizeof(int));
            hipMalloc(&devPtr2[j], fetch_num * sizeof(int));
            hipMalloc(&devPtr3[j], 2*fetch_num * sizeof(int));
            hipMemcpy(devPtr1[j], p_tmpbuffer1[j], fetch_num * sizeof(int), hipMemcpyHostToDevice);
            hipMemcpy(devPtr2[j], p_tmpbuffer2[j], fetch_num * sizeof(int), hipMemcpyHostToDevice);
        }
        int* devPtr4;
        int* devPtr5;
        int* devPtr6;
        hipMalloc(&devPtr4, partitial_num * sizeof(int));
        hipMalloc(&devPtr5, partitial_num * sizeof(int));
        hipMalloc(&devPtr6, partitial_num * sizeof(int));
        hipMemcpy(devPtr4, left_num_1, partitial_num * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(devPtr5, left_num_2, partitial_num * sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(devPtr6, destbuffer_num, partitial_num * sizeof(int), hipMemcpyHostToDevice);
        merge_kernel<<<partitial_num, fetch_num>>>(devPtr1, devPtr2, devPtr3, devPtr4, devPtr5, devPtr6);
        for(int j = 0; j < partitial_num; j++) {
            hipMemcpy(p_tmpbuffer1[j], devPtr1[j], fetch_num * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(p_tmpbuffer2[j], devPtr2[j], fetch_num * sizeof(int), hipMemcpyDeviceToHost);
            hipMemcpy(p_destbuffer[j], devPtr3[j], 2*fetch_num * sizeof(int), hipMemcpyDeviceToHost);
            hipFree(devPtr1[j]);
            hipFree(devPtr2[j]);
            hipFree(devPtr3[j]);
        }
        
        
        
        //* phase 3: write data to output file 
        for(int j = 0; j < partitial_num; j++) {
            if(destbuffer_num[j] > 0) {
                pwrite(out, p_destbuffer[j], destbuffer_num[j] * sizeof(int), offset + partitial_index1[j] * sizeof(int));
                partitial_index1[j] += destbuffer_num[j];
            }
        }
    }

}

void merge_two(int in, int out, int block_size, int thread_num, int offset1, int offset2, int entry_num1, int entry_num2) {
    // find partitial value in a
    int partitial_num = thread_num -1;
    int* partitial_value = (int*)malloc(partitial_num * sizeof(int));
    int* partitial_index1 = (int*)malloc(partitial_num * sizeof(int));
    int* partitial_index2 = (int*)malloc(partitial_num * sizeof(int));
    int* partitial_num1 = (int*)malloc(partitial_num * sizeof(int));
    int* partitial_num2 = (int*)malloc(partitial_num * sizeof(int));
    for(int i = 0; i < partitial_num; i++) {
        partitial_index1[i] = i * entry_num1 / partitial_num;
        partitial_value[i] = 0;
        pread(in, &partitial_value[i], sizeof(int), offset1 + partitial_index1[i] * sizeof(int));
        partitial_num1[i] = (i == partitial_num - 1) ? entry_num1 - partitial_index1[i] : partitial_index1[i+1] - partitial_index1[i];
    }
    // find partitial index in b
    binary_find_partition_index(in, partitial_num, partitial_value, offset2, entry_num2, partitial_index2, partitial_num2);
    // read data from input file

    // merge two sorted data in memory

}

void merge_pass(int in, int out, int block_num ,int thread_num, int * offset_info, int * entrynum_info, int block_size) {
    // read data from input file
    int merge_block_num = block_num  / 2;    //* if not the exponent of 2, the last block will be merged with the last block
    
    for(int i = 0; i < merge_block_num; i++) {
        merge_two(in, out, block_size, thread_num, offset_info[2*i], offset_info[2*i+1], entrynum_info[2*i], entrynum_info[2*i+1]);
    }
}


int merge_main(string fpath1, string fpath2, unsigned long long entry_num, unsigned long long block_size, int thread_num) {

    // phase 1: sort each block size data in memory 

    int fd_1;
    fd_1 = open(fpath1.c_str(), O_RDWR, 0);
    int fd_2;
    fd_2 = open(fpath2.c_str(),O_RDWR, 0);
    int block_num = (entry_num + block_size -1 )/block_size;
    for(int i = 0; i < block_num; i++) {
        if(i == block_num - 1) {
            block_sort(fd_1, i * block_size * sizeof(int), entry_num - i * block_size, thread_num);
        }
        else {
            block_sort(fd_1, i * block_size * sizeof(int), block_size, thread_num);
        }
    }
    
    //phase 2: merge sorted data
    int offset_info_even[block_num];
    int offset_info_odd[block_num];
    int entrynum_info_even[block_num];
    int entrynum_info_odd[block_num];
    for(int i = 0; i < block_num -1; i++) {
        offset_info_even[i] = i * block_size * sizeof(int);
        entrynum_info_even[i] = block_size;
    }
    offset_info_even[block_num - 1] = (block_num - 1) * block_size * sizeof(int);
    entrynum_info_even[block_num - 1] = entry_num - (block_num - 1) * block_size;
    int block_num_even = block_num;
    int block_num_odd ;
    //int block_size; //todo need to be calculated
    int pass_num = floor(log2(block_num));
    for(int i=0; i<pass_num; i++) {
        if(i % 2 == 0) {
            block_num_odd = block_num_even / 2;
            merge_pass(fd_1, fd_2, block_num_even, thread_num, offset_info_even, entrynum_info_even,block_size);
        }
        else {
            block_num_even = block_num_odd / 2;
            merge_pass(fd_2, fd_1, block_num_odd, thread_num, offset_info_odd, entrynum_info_odd,block_size);
        }
    }
    close(fd_1);
    close(fd_2);
    return 0;
}



int main(void)
{
    string s1="/home/szy/ssd1/testfile1";
    string s2="/home/szy/ssd1/testfile2";
    merge_main(s1,s2,4UL*1024*1024*1024,2UL*1024*1024*1024,128);
    int fd = open(s1.c_str(), O_RDWR, 0);
    check_result(fd,4UL*1024*1024*1024);
    return 0;
}